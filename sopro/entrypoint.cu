#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2>   result_buffer;

rtDeclareVariable(float4,                color, , );

RT_PROGRAM void drawColor()
{
  result_buffer[launch_index] = color;
}
