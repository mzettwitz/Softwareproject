#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "../cuda/helpers.h"
#include "../include/structs.h"
#include "../cuda/random.h"

using namespace optix;

//ray from camera to objects, returned payload to buffer

//camera parameters
rtDeclareVariable(float3, eye,,);
rtDeclareVariable(float3,U,,);
rtDeclareVariable(float3,V,,);
rtDeclareVariable(float3,W,,);

//color for exceptions
rtDeclareVariable(float4,exceptionColor,,);
//min distance
rtDeclareVariable(float,sceneEpsilon,,);
//output buffer
rtBuffer<uchar4,2> outputBuffer;
// 'head' of Scenetree
rtDeclareVariable(rtObject, topObject,,);
//which kind of ray
rtDeclareVariable(unsigned int, radianceRayType,,);
//output image size
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);
rtDeclareVariable(uint2, launchDim, rtLaunchDim,);




//perspective view
RT_PROGRAM void pinholeCamera()
{

    //setup camera, shift over every pixel
    float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2.f - 1.f;
    float3 rayOrigin = eye;
    float3 direction;
    optix::Ray ray;


        direction= normalize((d.x) * U + (d.y) * V + W);

        ray = optix::make_Ray(rayOrigin,direction,radianceRayType,sceneEpsilon,RT_DEFAULT_MAX);


    //create ray

    //trace radiance 'normal' rays from camera into scene
    PerRayData_radiance prd;

    prd.importance = 1.f;
    prd.depth = 0;

    float4 result = make_float4(0,0,0,0);

    rtTrace(topObject, ray, prd);
    result += prd.result;


    //return color to outputBuffer for each pixel
    outputBuffer[launchIndex] = make_color(make_float3(result.x,result.y,result.z));
}

//if exception return excpetionColor as payload
RT_PROGRAM void exception()
{
    const unsigned int code = rtGetExceptionCode();
    rtPrintf("Caugt Exception 0x%X at launch index(%d,%d)\n", code, launchIndex.x,launchIndex.y);
    outputBuffer[launchIndex] = make_color(make_float3(exceptionColor.x,exceptionColor.y,exceptionColor.z));
}
