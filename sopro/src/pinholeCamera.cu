#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "../cuda/helpers.h"

using namespace optix;

//ray from camera to objects, returned payload to buffer
struct PerRayData_radiance
{
    //color
    float3 result;
    //not used now
    float importance;
    //recursion depth
    int depth;
};


//camera parameters
rtDeclareVariable(float3, eye,,);
rtDeclareVariable(float3,U,,);
rtDeclareVariable(float3,V,,);
rtDeclareVariable(float3,W,,);

//color for exceptions
rtDeclareVariable(float3,exceptionColor,,);
//min distance
rtDeclareVariable(float,sceneEpsilon,,);
//output buffer
rtBuffer<uchar4,2> output_buffer;
// 'head' of Scenetree
rtDeclareVariable(rtObject, topObject,,);
//which kind of ray
rtDeclareVariable(unsigned int, radiance_ray_type,,);
//output image size
rtDeclareVariable(uint2, launchIndex, rtLauchindex,);
rtDeclareVariable(uint2, launchDim, rtLaunchDim,);


//perspective view
RT_PROGRAM void pinholeCamera()
{
    //setup camera, shift over every pixel
    float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2.f - 1.f;
    float3 rayOrigin = eye;
    float3 rayDirection = normalize(d.x * U + d.y * V + W);
    //create ray
    optix::Ray ray = optix::make_Ray(rayOrigin,rayDirection,radiance_ray_type,sceneEpsilon,RT_DEFAULT_MAX);
    //trace radiance 'normal' rays from camera into scene
    PerRayData_radiance prd;

    prd.importance = 1.f;
    prd.depth = 0;
	
    rtTrace(topObject, ray, prd);
	//return color to output_buffer for each pixel
    output_buffer[launchIndex] = make_color(prd.result);
}

//if exception return excpetionColor as payload
RT_PROGRAM void exception()
{
    const unsigned int code = rtGetExceptionCode();
    rtPrintf("Caugt Exception 0x%X at launch index(%d,%d)\n", code, launchIndex.x,launchIndex.y);
    output_buffer[launchIndex] = make_color(exceptionColor);
}
