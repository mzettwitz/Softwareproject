#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

//ray from camera to objects
struct PerRayData_radiance
{
    //color
    float4 result;
    //not used now
    float importance;
    //recursion depth
    int depth;
};


//camera parameters
rtDeclareVariable(float4, eye,,);
rtDeclareVariable(float4,U,,);
rtDeclareVariable(float4,V,,);
rtDeclareVariable(float4,W,,);


rtDeclareVariable(float4,exceptionColor,,);
rtDeclareVariable(float,sceneEpsilon,,);
rtBuffer<float4,2> output_buffer;
