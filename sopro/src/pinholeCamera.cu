#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

//ray from camera to objects
struct PerRayData_radiance
{
    //color
    float4 result;
    //not used now
    float importance;
    //recursion depth
    int depth;
};


//camera parameters
rtDeclareVariable(float3, eye,,);
rtDeclareVariable(float3,U,,);
rtDeclareVariable(float3,V,,);
rtDeclareVariable(float3,W,,);


rtDeclareVariable(float3,exceptionColor,,);
rtDeclareVariable(float,sceneEpsilon,,);
rtBuffer<float3,2> output_buffer;
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, radiance_ray_type,,);

rtDeclareVariable(uint2, launchIndex, rtLauchindex,);
rtDeclareVariable(uint2, launchDim, rtLaunchDim,);



RT_PROGRAM void pinholeCamera()
{
    float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2.f - 1.f;
    float3 rayOrigin = eye;
    float3 rayDirection = normalize(d.x * U, d.y * V + W);

    optix::Ray ray = optix::make_Ray(rayOrigin,rayDirection,radiance_ray_type,sceneEpsilon,RT_DEFAULT_MAX);

    PerRayData_radiance prd;

    prd.importance = 1.f;
    prd.depth = 0;

    rtTrace(topObject, ray, prd);

    output_buffer[launchIndex] = make_color(prd.result);
}


RT_PROGRAM void exception()
{
    const unsigned int code = rtGetExceptionCode();
    rtPrintf("Caugt Exception 0x%X at launch index(%d,%d)\n", code, launchIndex.x,launchIndex.y);
    output_buffer[launchIndex] = make_color(exceptionColor);
}
