#include "hip/hip_runtime.h"
#include "sopro/include/light.h"

Light::Light(){
    this->color = float4(1,1,1,1);
    this->origin = float4(0,0,0,0);
}

Light::Light(float4 ori, float4 col){
    this->origin = ori;
    this->color = col;
}

Light::~Light(){}

void Light::setOrigin(float4 newOrigin){
    this->origin = newOrigin;
}

void Light::setColor(float4 newColor){
    this->color = newColor;
}

float4 Light::getOrigin(){
    return this->origin;
}

float4  Light::getColor(){
    return this->color;
}
