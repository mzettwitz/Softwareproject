#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */



#include "../include/material.h"



using namespace optix;

struct PerRayData_shadow()
{
	float3 attenuation;
};

struct PerRayData_radiance()
{
	float3 result;
	float importance;
	int depth;
};

Material::Material(){
    this->color = make_float4(0,0,0,1);
}

Material::Material(float4 color){
    this->color = color;
}

Material::~Material(){}

void Material::setColor(float4 newColor){
    this->color = newColor;
}

float4 Material::getColor(){
    return this->color;
}


rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(optix::Ray ray,rtCurrentRay,);


static __device__ void shadowed()
{
	//create ray from hit to light
	prd_shadow.attenuation = make_float3(0,0,0);
	rtTerminateRay();
}

static __device__ void shade()
{
}
