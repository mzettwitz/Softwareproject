#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */

#include "../include/material.h"

using namespace optix;


RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}


/*! \fn shadowed
 * \brief method to generate a shadow
 *
 * Generate a new shadow ray if a the incoming ray hits an object
 */

static __device__ void shadowed()
{
    //create ray from hit to light
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}


/*! * \fn shade
 * \brief shade method for ray color
 * \var shadowPrd Shadow ray data (attenuation)
 * \var hitPoint Intersection coords with object
 * \var shadowDriection 3d vector of the shadow direction
 * \var shadowRay New ray to trace the shadow
 *
 * generate color, trace new shadow ray
 */

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    for(unsigned int i = 0;i < lights.size();++i)
    {

        float3 hitPoint = ray.origin + intersectionDistance * ray.direction;
        float3 shadowDirection = normalize(lights[i].pos - hitPoint);
        hitPoint = hitPoint + sceneEpsilon * shadowDirection;
        float maxLambda = length(lights[i].pos - hitPoint);

        Ray shadowRay = make_Ray(hitPoint, shadowDirection,
                                 shadowRayType, sceneEpsilon,maxLambda);

        //trace new shadow ray
        rtTrace(topShadower, shadowRay, shadowPrd);
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            result += make_float4(color,1.f) * make_float4(lights[i].color,1.f);
        }
        else
        {
            result += make_float4(0.0f,0.0f,0.0f,1.0f);
        }
    }
    result.w = 1.0f;
    prd_radiance.result = result;
}

