#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */

#include "../include/material.h"

using namespace optix;

/*! \fn shadowed
 * \brief method to generate a shadow
 *
 * generate a new shadow ray if a the ray hits an object
 */

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    //create ray from hit to light
    prd_shadow.attenuation = make_float3(0,0,0);
    rtTerminateRay();
}


/*! * \fn shade
 * \brief shade method ray color
 * \var shadowPrd shadow ray data
 * \var hitPoint intersection coords with object
 * \var shadowDriection 3d vector of the shadow direction
 * \var shadowRay is a new ray to trace the shadow
 *
 * generate color, trace new shadow ray
 */
static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;
    float3 shadowDirection = lights[0].pos - hitPoint;
    shadowDirection = normalize(shadowDirection);

    Ray shadowRay(hitPoint, shadowDirection,
                  shadowType, sceneEpsilon);

    //trace new shadow ray
    rtTrace(topShadower, shadowRay, shadowPrd);

    prd_radiance.result = color*shadowPrd.attenuation;

}

