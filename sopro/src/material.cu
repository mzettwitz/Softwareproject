#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */

#include "../include/material.h"

using namespace optix;


RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}


/*! \fn shadowed
 * \brief method to generate a shadow
 *
 * Generate a new shadow ray if a the incoming ray hits an object
 */

static __device__ void shadowed()
{
    //create ray from hit to light
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}


/*! * \fn shade
 * \brief shade method for ray color
 * \var shadowPrd Shadow ray data (attenuation)
 * \var hitPoint Intersection coords with object
 * \var shadowDriection 3d vector of the shadow direction
 * \var shadowRay New ray to trace the shadow
 *
 * generate color, trace new shadow ray
 */

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;
    float3 shadowDirection = normalize(lights[0].pos - hitPoint);
    hitPoint = hitPoint + sceneEpsilon * shadowDirection;
    float maxLambda = length(lights[0].pos - hitPoint);

    Ray shadowRay = make_Ray(hitPoint, shadowDirection,
                  shadowRayType, sceneEpsilon,2 * maxLambda);

    //trace new shadow ray
    rtTrace(topShadower, shadowRay, shadowPrd);
    if(fmaxf(shadowPrd.attenuation) > 0.0f)
    {
        prd_radiance.result = make_float4(color,1.f) * make_float4(lights[0].color,1.f);
    }
    else
    {
        prd_radiance.result = make_float4(0.0f,0.0f,0.0f,1.0f);
    }

}

