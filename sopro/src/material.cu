#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */

#include "sopro/include/material.h"

Material::Material(){
    this->color = float4(0,0,0,1);
}

Material::Material(float4 color){
    this->color = color;
}

Material::~Material(){}

void Material::setColor(float4 newColor){
    this->color = newColor;
}

float4 Material::getColor(){
    return this->color;
}
