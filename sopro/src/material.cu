#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */

#include "../include/material.h"

using namespace optix;

/*! \fn shadowed
 * \brief method to generate a shadow
 *
 * generate a new shadow ray if a the ray hits an object
 */

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    //create ray from hit to light
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}


/*! * \fn shade
 * \brief shade method ray color
 * \var shadowPrd shadow ray data
 * \var hitPoint intersection coords with object
 * \var shadowDriection 3d vector of the shadow direction
 * \var shadowRay is a new ray to trace the shadow
 *
 * generate color, trace new shadow ray
 */
static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;
    float3 shadowDirection = lights[0].pos - hitPoint;
    shadowDirection = normalize(shadowDirection);

    Ray shadowRay(hitPoint, shadowDirection,
                  shadowRayType, sceneEpsilon);

    //trace new shadow ray
    rtTrace(topShadower, shadowRay, shadowPrd);
    if(fmaxf(shadowPrd.attenuation) > 0.0f)
    {
        prd_radiance.result = make_float4(color,1.f);
    }
    else
    {
        prd_radiance.result = make_float4(0.0f,0.0f,0.0f,1.0f);
    }

}

