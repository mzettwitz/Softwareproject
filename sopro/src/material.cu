#include "hip/hip_runtime.h"
/*!
 * TODO: closest hit
 * anyhit, for shadows
 *
 */



#include "../include/material.h"

using namespace optix;

Material::Material(){
    this->color = make_float4(0,0,0,1);
}

Material::Material(float4 color){
    this->color = color;
}

Material::~Material(){}

void Material::setColor(float4 newColor){
    this->color = newColor;
}

float4 Material::getColor(){
    return this->color;
}
