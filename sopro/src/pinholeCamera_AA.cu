#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../include/random.h"
#include "../include/structs.h"
#include "../include/helpers.h"

using namespace optix;

rtDeclareVariable(unsigned int,  frameNumber, , );
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        exceptionColor, , );
rtDeclareVariable(float,         sceneEpsilon, , );
rtBuffer<uchar4, 2>              outputBuffer;
rtBuffer<float4, 2>              varianceSumBuffer;
rtBuffer<float4, 2>              varianceSum2Buffer;
rtBuffer<unsigned int, 2>        numSamplesBuffer;
rtBuffer<unsigned int, 2>        randomSeeds;
rtDeclareVariable(rtObject,      topObject, , );
rtDeclareVariable(unsigned int,  radianceRayType, , );

rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

// Trace ray through screen_coord
static __device__ __inline__ float3 trace( float2 screen_coord )
{
  size_t2 screen = outputBuffer.size();
  float2 d = screen_coord / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radianceRayType, sceneEpsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(topObject, ray, prd);
  return make_float3(prd.result.x,prd.result.y,prd.result.z);
}


static __device__ __inline__ float3 jittered_trace( const uint2& index )
{
    // Trace a randomly offset ray within the pixel
    volatile unsigned int seed  = randomSeeds[ index ]; // volatile workaround for cuda 2.0 bug
    unsigned int new_seed  = seed;
    float uu = rnd( new_seed )-0.5f;
    float vv = rnd( new_seed )-0.5f;
    randomSeeds[ launch_index ] = new_seed;

    float2 offset = make_float2( uu, vv );
    float3 result = trace( offset + make_float2( index ) );

    return result;
}

RT_PROGRAM void pinholeCamera()
{
  if ( frameNumber == 0 ) {
    float3 result = jittered_trace( launch_index );
    outputBuffer[ launch_index ] = make_color( result );

    // Update buffers
    numSamplesBuffer[ launch_index ]   = 1u;
    varianceSumBuffer[ launch_index ]  = make_float4(result, 0.0f);
    varianceSum2Buffer[ launch_index ] = make_float4(result*result, 0.0f);
  } else {
    {
      // ns < 0x80000000 means the variance is too high and we should keep rendering.
      volatile unsigned int ns = numSamplesBuffer[ launch_index ];
      if ( (ns & 0x80000000) && (((launch_index.y >> 3) & 0x3) != (frameNumber & 0x3)) ) {
        return;
      }
    }

    float3 new_color = jittered_trace( launch_index );

    // Add in new ray's contribution
    volatile unsigned int ns = numSamplesBuffer[ launch_index ] & ~0x80000000; // volatile workaround for Cuda 2.0 bug
    float  new_value_weight = 1.0f / (float)ns;
    float  old_value_weight = 1.0f - new_value_weight;
    uchar4& old_bytes = outputBuffer[ launch_index ];
    float3 old_color = make_float3(old_bytes.z, old_bytes.y, old_bytes.x)*make_float3(1.f/255.0f);
    float3 result = old_color*old_value_weight + new_color*new_value_weight;

    // Update buffers
    outputBuffer[ launch_index ] = make_color(result);
    float4 vsum  = varianceSumBuffer[ launch_index ];
    float4 vsum2 = varianceSum2Buffer[ launch_index ];
    // Compute the variance of the series of displayed pixels over time. This variance will go to zero, regardless of the variance of the sample values.
    varianceSumBuffer[ launch_index ]  = vsum  = vsum  + make_float4( result, 0.0f );
    varianceSum2Buffer[ launch_index ] = vsum2 = vsum2 + make_float4( result*result, 0.0f );
    ns++;

    // If we are beyond our first four samples per pixel, check variance
    if ( frameNumber > 6 ) {
      float3 rgb_variance = ( make_float3( vsum2 ) - make_float3( vsum ) * make_float3( vsum ) * new_value_weight ) * new_value_weight;

      float variance = optix::luminance( rgb_variance );
      // render an 8-row span every 32 rows regardless. This shape lets entire warps turn off.
      if ( variance < 0.001f ) {
        ns = ns | 0x80000000;
      }
    }

    numSamplesBuffer[ launch_index ] = ns;
  }
}


RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  outputBuffer[launch_index] = make_color(exceptionColor);
}
