#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3>    vertexBuffer;
rtBuffer<float3>    normalBuffer;
//rtBuffer<float2>    uvBuffer;
rtBuffer<int3>      vertexIndexBuffer;
rtBuffer<int3>      normalIndexBuffer;
//rtBuffer<int3>      uvIndexBuffer;

//rtDeclareVariable(float3,texCoord, attribute texCoord,);

rtDeclareVariable(float3,   geometricNormal,    attribute geometricNormal,);
rtDeclareVariable(float3,   shadingNormal,      attribute shadingNormal,);
rtDeclareVariable(Ray,      ray,                rtCurrentRay,);

RT_PROGRAM void meshIntersectionProgram(int primIdx)
{
    //get triangle
    int3 vertexIndex = vertexIndexBuffer[primIdx];

    float3 p0 = vertexBuffer[vertexIndex.x];
    float3 p1 = vertexBuffer[vertexIndex.y];
    float3 p2 = vertexBuffer[vertexIndex.z];

    float3 normal;
    float t, beta, gamma;
    //check for intersection
    if(intersect_triangle(ray,p0,p1,p2,normal,t,beta,gamma))
    {
        if(rtPotentialIntersection(t))
        {
            //get normal
            int3 normalIndex = normalIndexBuffer[primIdx];
            //interpolate if normals given

            if(normalBuffer.size() == 0 || normalIndex.x < 0 || normalIndex.y < 0 || normalIndex.z < 0)
            {
                //no normals given, use calculated from intersection
                shadingNormal = normalize(normal);
            }
            else
            {
                float3 n0 = normalBuffer[normalIndex.x];
                float3 n1 = normalBuffer[normalIndex.y];
                float3 n2 = normalBuffer[normalIndex.z];
                shadingNormal = normalize(n1*beta + n2*gamma + n0*(1.0f-beta-gamma));
            }
            geometricNormal = normalize(normal);

           /* //check for texture
            int3 uvIndex = uvIndexBuffer[primIdx];
            if(uvBuffer.size() == 0 || uvIndex.x < 0 || uvIndex.y < 0 || uvIndex.z < 0)
            {
                //no textureCoordinates given
                texCoord = make_float3(0.0f,0.0f,0.0f);
            }
            else
            {
                float t0 = uvBuffer[uvIndex.x];
                float t1 = uvBuffer[uvIndex.y];
                float t2 = uvBuffer[uvIndex.z];
                texCoord = make_float3(t1*beta+t2*gamma * t0(1.0f-beta-gamma));
            }*/

            rtReportIntersection(0);

        }
    }
}

RT_PROGRAM void meshBoundingBoxProgram(int primIdx, float result[6])
{
    const int3 vertexIndex = vertexIndexBuffer[primIdx];

    const float3 v0 = vertexBuffer[vertexIndex.x];
    const float3 v1 = vertexBuffer[vertexIndex.y];
    const float3 v2 = vertexBuffer[vertexIndex.z];

    const float area = length(cross(v1-v0,v2-v0));

    Aabb* aabb = (optix::Aabb*)result;

    if(area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0,v1),v2);
        aabb->m_max = fmaxf(fmaxf(v0,v1),v2);
    }
    else
    {
        aabb->invalidate();
    }
}
