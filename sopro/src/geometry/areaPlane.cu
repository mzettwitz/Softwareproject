#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(float3, position,,);
rtDeclareVariable(float3,dimensions,,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(float,sceneEpsilon,,);
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal,);
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal,);


RT_PROGRAM void areaPlaneIntersectionProgram(int primIdx)
{

}


RT_PROGRAM void areaPlaneBoundingBoxProgram(int primIdx, float result[6])
{

}
