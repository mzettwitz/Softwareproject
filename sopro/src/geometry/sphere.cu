#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>


using namespace optix;

rtDeclareVariable(float4,   coordinates,                ,);
rtDeclareVariable(Ray,      ray,        rtCurrentRay    ,);
rtDeclareVariable(float3,   normal,     attribute normal,);

RT_PROGRAM void sphereIntersectionProgram(int primIdx)
{

    //to solve : (o + td - c)² - r² = 0
    // t_1,2 = - (od - dc)/d² +- sqrt(((od-dc)/d²)² - (2oc + o² + c² -r²)/d²

    float3 c        = make_float3(coordinates);
    float3 o        = ray.origin;
    float3 d        = ray.direction;
    float  r        = coordinates.w;

    float  b        = (dot(o,d) - dot(d,c)) / dot(d,d);
    float  e        = (-2 * dot(o,c) + dot(o,o) + dot(c,c) - r*r)/dot(d,d);
    float  disc     = b*b - e;

    if(disc >= 0.0f)
    {
        disc = sqrtf(disc);
        float lambda1 = -b-disc;
        float lambda2 = -b+disc;

        if(lambda1 < lambda2 && lambda1 > 0.0f)
        {
            if(rtPotentialIntersection(lambda1))
            {
                normal = -c + o + lambda1 * d;
                rtReportIntersection(0);
            }
        }
        else
        {
            if(rtPotentialIntersection(lambda2))
            {
                normal =  -c + o + lambda2 * d;
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void sphereBoundingBoxProgram(int,float result[6])
{
    //use aabb(axis aligned bounding box)
    //just set min and max coordinates

    float3 c = make_float3(coordinates);
    float3 r = make_float3(coordinates.w);

    Aabb* aabb = (Aabb*)result;

    if(r.x > 0.0f && !isinf(r.x))
    {
        aabb->m_min = c - r;
        aabb->m_max = c + r;
    }
    else
    {
        aabb->invalidate();
    }
}
