#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>


using namespace optix;

rtDeclareVariable(float3,   coordinates,                ,);
rtDeclareVariable(float,    radius,                     ,);
rtDeclareVariable(Ray,      ray,        rtCurrentRay    ,);
rtDeclareVariable(float3,   geometricNormal, attribute geometricNormal,);
rtDeclareVariable(float3,   shadingNormal, attribute shadingNormal,);


RT_PROGRAM void sphereIntersectionProgram(int primIdx)
{

    //to solve : (o + td - c)² - r² = 0
    // t_1,2 = - (od - dc)/d² +- sqrt(((od-dc)/d²)² - (2oc + o² + c² -r²)/d²

    float3 c        = coordinates;
    float3 o        = ray.origin;
    float3 d        = ray.direction;
    float  r        = radius;

    float  b        = (dot(o,d) - dot(d,c)) / dot(d,d);
    float  e        = (-2 * dot(o,c) + dot(o,o) + dot(c,c) - r*r)/dot(d,d);
    float  disc     = b*b - e;

    if(disc >= 0.0f)
    {
        disc = sqrtf(disc);
        float lambda1 = -b-disc;
        float lambda2 = -b+disc;

        if(lambda1 < lambda2 && lambda1 > 0.0f)
        {
            if(rtPotentialIntersection(lambda1))
            {
                float3 normal = -c + o + lambda1 * d;
                normal = normalize(normal);
                geometricNormal = normal;
                shadingNormal = normal;
                rtReportIntersection(0);
            }
        }
        else
        {
            if(rtPotentialIntersection(lambda2))
            {
                float3 normal =  -c + o + lambda2 * d;
                normal = normalize(normal);
                geometricNormal = normal;
                shadingNormal = normal;
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void sphereBoundingBoxProgram(int,float result[6])
{
    //use aabb(axis aligned bounding box)
    //just set min and max coordinates

    float3 c = coordinates;
    float3 r = make_float3(radius);

    Aabb* aabb = (Aabb*)result;

    if(r.x > 0.0f && !isinf(r.x))
    {
        aabb->m_min = c - r;
        aabb->m_max = c + r;
    }
    else
    {
        aabb->invalidate();
    }
}
