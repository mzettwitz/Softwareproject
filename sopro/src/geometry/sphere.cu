#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>


using namespace optix;

rtDeclareVariable(Ray,      ray,        rtCurrentRay    ,);
rtDeclareVariable(float3,   geometricNormal, attribute geometricNormal,);
rtDeclareVariable(float3,   shadingNormal, attribute shadingNormal,);


RT_PROGRAM void sphereIntersectionProgram(int primIdx)
{

    //to solve : (o + td)² - 1 = 0
    // t_1,2 = - (od)/d² +- sqrt(((od)/d²)² - (o²-1)/d²
    float3 o        = ray.origin;
    float3 d        = ray.direction;

    float  b        = (dot(o,d)) / dot(d,d);
    float  e        = (dot(o,o) - 1)/dot(d,d);
    float  disc     = b*b - e;

    if(disc >= 0.0f)
    {
        disc = sqrtf(disc);
        float lambda1 = -b-disc;
        float lambda2 = -b+disc;

        if(lambda1 < lambda2 && lambda1 > 0.0f)
        {
            if(rtPotentialIntersection(lambda1))
            {
                float3 normal = o + lambda1 * d;
                normal = normalize(normal);
                geometricNormal = normal;
                shadingNormal = normal;
                rtReportIntersection(0);
            }
        }
        else
        {
            if(rtPotentialIntersection(lambda2))
            {
                float3 normal = o + lambda2 * d;
                normal = normalize(normal);
                geometricNormal = normal;
                shadingNormal = normal;
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void sphereBoundingBoxProgram(int,float result[6])
{
    //use aabb(axis aligned bounding box)
    //just set min and max coordinates

    Aabb* aabb = (Aabb*)result;
        aabb->m_min = make_float3(-1);
        aabb->m_max = make_float3(1);
}
