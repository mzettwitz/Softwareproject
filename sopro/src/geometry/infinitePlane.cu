#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>



using namespace optix;

rtDeclareVariable(float3,plane,,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(float,sceneEpsilon,,);
rtDeclareVariable(float3,shadingNormal, attribute shadingNormal,);
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal,);

RT_PROGRAM void infinitePlaneIntersectionProgram(int primIdx)
{
    float t = 0;
    if( t = (-ray.origin.y + plane.y) / ray.direction.y)
    {

        if(rtPotentialIntersection(t))
        {
            rtReportIntersection(0);
            shadingNormal = geometricNormal = make_float3(0.0f,1.0f,0.0f);
        }
    }
}

RT_PROGRAM void infinitePlaneBoundingBoxProgram(int primIdx, float result[6])
{
    result[0] = plane.x;
    result[1] = plane.y - sceneEpsilon;
    result[2] = plane.x;
    result[3] = plane.z;
    result[4] = plane.y + sceneEpsilon;
    result[5] = plane.z;
}
