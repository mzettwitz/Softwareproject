#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>



using namespace optix;

rtDeclareVariable(float3,plane,,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(float,sceneEpsilon,,);

RT_PROGRAM void infinitePlaneIntersectionProgram(int primIdx)
{
    float t = (-ray.origin.y + plane.y) / ray.direction.y;

    if(rtPotentialIntersection(t))
    {
        rtReportIntersection(0);
    }
}

RT_PROGRAM void infinitePlaneBoundingBoxProgram(int primIdx, float result[6])
{
    result[0] = plane.x;
    result[1] = plane.y - sceneEpsilon;
    result[2] = plane.x;
    result[3] = plane.z;
    result[4] = plane.y + sceneEpsilon;
    result[5] = plane.z;
}
