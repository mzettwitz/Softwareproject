#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float4, coordinates,,);
rtDeclareVariable(optix::Ray, ray,rtCurrentRay,);

RT_PROGRAM void sphereIntersect(int primIdx)
{
    float3 c = make_float3(coordinates);
    float3 o = ray.origin - c;
    float3 d = ray.direction;
    float radius = coordinates.w;

    float b = dot(o,d);
    float e = dot(o,o) - radius * radius;
    float discriminant = b*b - e;

    if(discriminant > 0.0f)
    {

        discriminant = sqrtf(discriminant);
        float lambda1 = -b-discriminant;
        float lambda2 = -b-discriminant;

        if(lambda1 < lambda2)
        {
            if(rtPotentialIntersection(lambda1))
            {
                rtReportIntersection(0);
            }
        }
        else
        {
            if(rtPotentialIntersection(lambda2))
            {
                    rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void sphereBounds(int,float result[6])
{
	//use aabb

    float3 c = make_float3(coordinates);
    float3 r = make_float3(coordinates.w);

    Aabb* aabb = (Aabb*)result;

    if(r.x > 0.0f && !isinf(r.x))
    {
        aabb->m_min = c - r;
        aabb->m_max = c + r;
    }
    else
    {
        aabb->invalidate();
    }
}
