#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float4, coordinates,,);
rtDeclareVariable(optix::Ray, ray,currentRay,);

RT_PROGRAM void sphereIntersect(int primIdx)
{
    float3 c = make_float3(coordinates);
    float3 o = ray.origin - center;
    float3 d = ray.direction;
    float radius = sphere.w;

    float b = dot(o,d);
    float c = dot(o,o) - radius * radius;
    float discriminant = b*b - c;

    if(discriminant > 0.0f)
    {

        discrimant = sqrtf(discriminant);
        float lambda1 = -b-discriminant;
        float lambda2 = -b-discriminant;


    }
}

RT_PROGRAM void sphereBounds(float result[6])
{
	//use aabb

    float3 c = make_float3(coordinates);
    float3 r = sphere.w;

    Aabb aabb = (Aabb*)result;

    if(r > 0.0f && !isinf(r))
    {
        aabb->m_min = c - r;
        aabb->m_max = c + r;
    }
    else
    {
        aabb->invalidate();
    }
}
