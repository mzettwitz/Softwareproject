#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, sphereMin,,);
rtDeclareVariable(float3, sphereMax,,);
rtDeclareVariable(optix::Ray, ray,currentRay,);
rtDeclareVariable(float3,geometricNormal, attribute geometric_normal,);
rtDeclareVariable(float3,shadingNormal, attribute shading_normal,);

RT_PROGRAM void sphereIntersect(int)
{
	//solve quadratic equation
}

RT_PROGRAM void sphereBounds(int, float center, float radius)
{
	//use aabb
}
