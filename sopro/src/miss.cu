#include "hip/hip_runtime.h"
#include <optix_world.h>

//color returned if no object hit
rtDeclareVariable(float3,missColor,,);
//raypayload for normal rays, see pinholeCamera.cu
struct PerRayData_radiance
{	//returning color
	float3 result;
	//not used now
	float importance;
	//recursionDepth
	int depth;
};
/*!
 * declare payload
 */
rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);

/*!
 *\brief return missColor, if no object was hit
 */
RT_PROGRAM miss void()
{
	prd_radiance.result = missColor;
}
