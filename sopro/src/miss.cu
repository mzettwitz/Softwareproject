#include "hip/hip_runtime.h"
#include <optix_world.h>

rtDeclareVariable(float3,missColor,,);

struct PerRayData_radiance
{
	float3 result;
	float importance;
	int depth;
};

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);

RT_PROGRAM miss void()
{
	prd_radiance.result = missColor;
}
