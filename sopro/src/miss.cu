#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "../include/structs.h"
//color returned if no object hit
rtDeclareVariable(float4,missColor,,);
//raypayload for normal rays, see pinholeCamera.cu

/*!
 * declare payload
 */
rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);

/*!
 *\brief return missColor, if no object was hit
 */
RT_PROGRAM void miss ()
{
	prd_radiance.result = missColor;
}
