#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "../include/structs.h"


//raypayload for normal rays, see pinholeCamera.cu


/*!
 *\brief return missColor
 * \var prd_radiance Information about incoming ray
 * \var missColor RGBA background color
 *
 * Set missColor to ray if no object was hitten by incoming ray
 */


rtDeclareVariable(float4,missColor,,);
rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);

RT_PROGRAM void miss ()
{
	prd_radiance.result = missColor;
}
