#include "hip/hip_runtime.h"
//TODO: everything

#include "../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(unsigned int,maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, ambientCoefficient,,);
rtDeclareVariable(float, diffuseCoefficient,,);
rtDeclareVariable(float, specularCoefficient,,);
rtDeclareVariable(float, shininess,,);
rtDeclareVariable(float, specularity,,);
rtDeclareVariable(float3, normal, attribute normal,);

static __device__ void shadowed();
static __device__ void shade();


RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);

    //color information
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    float3 diffuseColor = make_float3(0.0f,0.0f,0.0f);
    float3 ambientColor = make_float3(0.0f,0.0f,0.0f);
    float specularColor = 0;
    float3 phong = make_float3(0.0f,0.0f,0.0f);

    //hitpoint information
    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    //iterate through lights
    for(unsigned int i = 0;i < lights.size();++i)
    {
        //light values
        float3 lightDirection = lights[i].position - hitPoint;        
        float maxLambda = length(lightDirection);
        float radiance = lights[i].intensity / (maxLambda*maxLambda);
        lightDirection = normalize(lightDirection);
        float3 reflectedRay = reflect(lightDirection, normal);
        //reflectedRay = normalize(reflectedRay);

        // hitpoint offset
        hitPoint = hitPoint + sceneEpsilon * lightDirection;

        // trace new shadow ray
        Ray shadowRay = make_Ray(hitPoint,lightDirection,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        //phong = Ka + Kd + Ks
        //E = lights[i].intensity /dist²
        //Ka = ambientCoeff * ambientLightIntensity
        //Kd = diffuseCoeff * diffuseColor * distributionAngle * radiance(lightIntensity)
        //Ks = specularCoff * (shininess+2)/(2*PI)* (dot(ReflectedLight,ray.direction)^shininess * radiance


        // ambient outside to lighten shadowed parts
        ambientColor = lights[i].color * ambientCoefficient * radiance * color; ///-----------------------color??? correct?-------------///
        phong += ambientColor;

        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            diffuseColor = color * diffuseCoefficient * dot(normal, lightDirection) * radiance;

            //specularColor = specularCoefficient * (shininess + 2)/(2*M_PIf) * pow(dot(reflectedRay, ray.direction), shininess) * radiance;
            specularColor = specularCoefficient * ((shininess + 2)/(2*M_PIf)) * pow(dot(ray.direction, reflectedRay), shininess) * radiance;
            phong += diffuseColor;
            phong.x += specularColor;
            phong.y += specularColor;
            phong.z += specularColor;
        }
    }
/*
    //WILL NOT WORK!!!!
    if(specularity > 0.0f && prd_radiance.depth < maxDepth)
    {
        prd_radiance.depth++;
        float maxLambda = 10000;
        Ray reflectedRay = make_Ray(hitPoint,reflect(ray.direction,normal),radianceRayType,sceneEpsilon,maxLambda);
        //count depth + 1,
        rtTrace(topShadower,reflectedRay,prd_radiance);
        result = (1-specularity) * result + prd_radiance.result * specularity;
    }
*/

    result.x += phong.x;
    result.y += phong.y;
    result.z += phong.z;

    result.w = 1.0f;
    prd_radiance.result = result/lights.size();


}
