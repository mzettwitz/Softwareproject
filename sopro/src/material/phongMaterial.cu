#include "hip/hip_runtime.h"
//TODO: everything

#include "../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(unsigned int,maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, ambientCoefficient,,);
rtDeclareVariable(float, diffuseCoefficient,,);
rtDeclareVariable(float, specularCoefficient,,);
rtDeclareVariable(float, shininess,,);
rtDeclareVariable(float, specularity,,);
rtDeclareVariable(float3, normal, attribute normal,);

static __device__ void shadowed();
static __device__ void shade();


RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);

    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;





    for(unsigned int i = 0;i < lights.size();++i)
    {

        float3 lightDirection = lights[i].position - hitPoint;
        float maxLambda = length(lightDirection);
        lightDirection = normalize(lightDirection);
        hitPoint = hitPoint + sceneEpsilon * lightDirection;
        Ray shadowRay = make_Ray(hitPoint,lightDirection,shadowRayType,sceneEpsilon,maxLambda);

        rtTrace(topShadower,shadowRay,shadowPrd);

        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            //phong = Ka + Kd + Ks
            //E = lights[i].intensity * dot(normal.lightDirection)/dist²
            //Ka = ambientColor
            //Kd = E * (diffuseColor + diffuseCoefficient)
            //Ks = E * (dot(R,ray.direction)^shininess + specularCoefficient)
            //something like that

        }
    }
    //WILL NOT WORK!!!!
    if(specularity > 0.0f && prd_radiance.depth < maxDepth)
    {
        prd_radiance.depth++;
        float maxLambda = 10000;
        Ray reflectedRay = make_Ray(hitPoint,reflect(ray.direction,normal),radianceRayType,sceneEpsilon,maxLambda);
        //count depth + 1,
        rtTrace(topShadower,reflectedRay,prd_radiance);
        result = (1-specularity) * result + prd_radiance.result * specularity;
    }
}
