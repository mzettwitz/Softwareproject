#include "hip/hip_runtime.h"
//TODO: fix the shading to the brdf

#include "../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int,maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, ambientCoefficient,,);
rtDeclareVariable(float, diffuseCoefficient,,);
rtDeclareVariable(float, specularCoefficient,,);
rtDeclareVariable(float, shininess,,);
rtDeclareVariable(float, specularity,,);
rtDeclareVariable(float3, normal, attribute normal,);

static __device__ void shadowed();
static __device__ void shade();

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not using \fn shadowed.
 */
RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

/*!
 * \brief Computes the closest intersection between camera ray and the \class SceneObject using \fn shade.
 */
RT_PROGRAM void closesthit_radiance()
{
    shade();
}

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not, sets the attenuation to 0 and terminates the ray.
 */
static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

/*!
 * \brief Computes the color of the closest intersection point from camera and object.
 *
 * Basicly we set up a resulting color. This color is basicly black, it gains more color information with each
 * computation for each light source in the scene. \n
 * First we compute the hitpoint on the object. Now we iterate through each lightsource in the scene
 * and generate a new \class Ray from lightsource to the object to trace the shadow (using anyhit).
 * If there is shadow: add black, if there is no shadow, compute the phong based color.
 * Therefore we sum up the ambient light, the diffuse color and the specular color and add it,
 * weighted by the number of lights in the scene, to the resulting color.\n
 * At the end we generate and trace a new reflected \class Ray to compute the mirrored objects in the scene.
 *
 * \var shadowPrd The ray information for the shadow ray.
 * \var result RGBA color for the resulting color in the hitpoint.
 * \var diffuseColor RGB color for the diffuse part of the \class PhongMaterial
 * \var ambientColor RGB color for the ambient light color of the \class PhongMaterial
 * \var specularColor Float value to add highlights to \class PhongMaterial
 * \var phong RGB color to sum up the three parts of the \class PhongMaterial
 * \var hitPoint 3D float vector for the intersection between camera ray and object
 * \var lightDirection 3D float vector for the direction of the actual lightsource
 * \var maxLambda Float value describing the distance between actual lightsource and hitpoint
 * \var reflectedLightRay 3D vector for the direction of the light ray, reflected on the object surface
 * \var shadowRay A \class Ray to determine if the hitpoint is shadowed by the actual lightsource and any object in the scene
 * \var reflectedRay A \class Ray to trace the camera \class Ray that is reflected on the specular surface (mirror effect)
 * \var color4F RGBA version of the RGB diffuse color to give the reflection a color(lighted areas must be colored, since light is emitted)
 *
 */
static __device__ void shade()
{
    PerRayData_shadow shadowPrd;


    //color information
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    float3 diffuseColor = make_float3(0.0f,0.0f,0.0f);
    float3 ambientColor = make_float3(0.0f,0.0f,0.0f);
    float3 specularColor = make_float3(0.0f,0.0f,0.0f);
    float3 phong = make_float3(0.0f,0.0f,0.0f);

    float3 worldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,normal));
    float3 ffNormal = faceforward(worldNormal,-ray.direction,worldNormal);

    //hitpoint information
    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    //iterate through lights
    for(unsigned int i = 0;i < lights.size();++i)
    {
        //must be set here, otherways, only 1! shadow will be interpreted the right way
        shadowPrd.attenuation = make_float3(1.0f);
        //light values
        float3 lightDirection = lights[i].position - hitPoint;        
        float maxLambda = length(lightDirection) + sceneEpsilon;
        float radiance = lights[i].intensity / (maxLambda*maxLambda);
        lightDirection = normalize(lightDirection);
        float3 reflectedLightRay = reflect(lightDirection, ffNormal);
        reflectedLightRay = normalize(reflectedLightRay);

        // hitpoint offset
        hitPoint = sceneEpsilon * ffNormal + hitPoint;
        // trace new shadow ray
        Ray shadowRay = make_Ray(hitPoint,lightDirection,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        //phong = Ka + Kd + Ks
        //radiance = lights[i].intensity /dist²
        //Ka = ambientCoeff * ambientLightIntensity
        //Kd = diffuseCoeff * diffuseColor * distributionAngle(surfaceNormal, light) * radiance(localLightIntensity)
        //Ks = specularCoff * (shininess+2)/(2*PI)* distributionAngle(ReflectedLight, eyeVector)^shininess * radiance


        // ambient outside to lighten shadowed parts
        ambientColor = lights[i].color * color *  ambientCoefficient * radiance;
        phong = ambientColor;

        // if not in shadow
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            // material color * coeff * (positive)surface angle * lightintensity at hitpoint
            diffuseColor = lights[i].color * color * diffuseCoefficient * dot(ffNormal, lightDirection) * radiance;
            // lightcolor * coeff * normalized shininess * (positive)angle between eye and reflected light ray ^ shininess * lightintensity at hitpoint
            specularColor = lights[i].color * specularCoefficient * ((shininess + 2.f)/(2.f*M_PIf)) *
                    pow(fmaxf(dot(ray.direction, reflectedLightRay),0.f), shininess) * radiance;

            phong += diffuseColor + specularColor;

        }
    }

    result.x += phong.x;
    result.y += phong.y;
    result.z += phong.z;

    // recursive reflections
    if(specularity > 0.0f && prd_radiance.depth < maxDepth)
    {

        PerRayData_radiance prd_radiance_reflect;
        prd_radiance_reflect.depth = prd_radiance.depth+1;

        float maxLambda = 10000.0f;
        Ray reflectedRay = make_Ray(hitPoint,reflect(ray.direction,normal),radianceRayType,sceneEpsilon,maxLambda);
        rtTrace(topObject, reflectedRay, prd_radiance_reflect);
        result = (1.0f-specularity) * result + prd_radiance_reflect.result * specularity;
    }

    result.w = 1.0f;

    prd_radiance.result = result/lights.size();


}
