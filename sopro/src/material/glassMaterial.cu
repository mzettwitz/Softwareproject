#include "hip/hip_runtime.h"
// TODO:
// - refraction method correct?
// - trace with stack
// - Beer's law

#include "../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(unsigned int,maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float4, color,,);
rtDeclareVariable(float, refractiveIdx,,);
rtDeclareVariable(float, specularity,,);
rtDeclareVariable(float, shininess,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3, normal, attribute normal,);

static __device__ void shadowed();
static __device__ void shade();
static __device__ float3 refract(const float3 &ray_in, const float3 &normal, float n1, float n2);

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not using \fn shadowed.
 */
RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

/*!
 * \brief Computes the closest intersection between camera ray and the \class SceneObject using \fn shade.
 */
RT_PROGRAM void closesthit_radiance()
{
    shade();
}

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not, sets the attenuation to 0 and terminates the ray.
 */
static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

/*!
 * \brief Computes the color of the closest intersection point from camera and object.
 *
 * Basicly we set up a resulting color. This color is basicly black, it gains more color information with each
 * computation for each light source in the scene. \n
 * First we compute the hitpoint on the object. Now we iterate through each lightsource in the scene
 * and generate a new \class Ray from lightsource to the object to trace the shadow (using anyhit).
 * If there is shadow: add black, if there is no shadow, compute the reflected and refracted color.
 * Therefore we reflect(generate) some rays to the outside and refract(generate) some rays to the inside. The refracted Ones depend on the
 * refractive index using Snell's law and Beer's law. \n
 *
 * \var shadowPrd The ray information for the shadow ray.
 * \var result RGBA color for the resulting color in the hitpoint.
 * \var reflectiveColor RGB color for reflective color information in the hitpoint
 * \var refractiveColor RGB color for the refractive color information in the hintpoint
 * \var glassColor RGB color for the sum of the reflection, refraction and the specular light distribution(phong based)
 * \var hitPoint 3D float vector for the intersection between camera ray and object
 * \var lightDirection 3D float vector for the direction of the actual lightsource
 * \var maxLambda Float value describing the distance between actual lightsource and hitpoint
 * \var reflectedLightRay 3D vector for the direction of the light ray, reflected on the object surface
 * \var refractedLightRay 3D vector for the direction of the light ray, refracted on the object surface
 * \var shadowRay A \class Ray to determine if the hitpoint is shadowed by the actual lightsource and any object in the scene
 * \var reflectedRay A \class Ray to trace the camera \class Ray that is reflected on the specular surface (mirror effect)
 * \var refractedRay A \class Ray to trace the camera \class Ray that is refracted on the transparent surface (refraction effect)
 *
 */
static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);

    //color information
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    float3 reflectiveColor = make_float3(0.0f,0.0f,0.0f);
    float3 refractiveColor = make_float3(0.0f,0.0f,0.0f);
    float3 glassColor = make_float3(0.0f,0.0f,0.0f);

    //hitpoint information
    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    //iterate through lights
    for(unsigned int i = 0;i < lights.size();++i)
    {
        //light values
        float3 lightDirection = lights[i].position - hitPoint;
        float maxLambda = length(lightDirection);
        float radiance = lights[i].intensity / (maxLambda*maxLambda);
        lightDirection = normalize(lightDirection);
        float3 reflectedLightRay = reflect(lightDirection, normal);
        reflectedLightRay = normalize(reflectedLightRay);

        // hitpoint offset
        hitPoint = hitPoint + sceneEpsilon * normal;

        // trace new shadow ray
        Ray shadowRay = make_Ray(hitPoint,lightDirection,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        result.x = color.x * color.w;
        result.y = color.y * color.w;
        result.z = color.z * color.w;

        // if not in shadow
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            // add highlights with phong based light distribution

            // recursive refraction
            /* TODO:

            should look like:
            - compute the refracted direction
            - put actual material on a stack
            - trace new refracted ray till it hit's something different from glass material or nothing
                -> recursion
            */

            // phong based highlights
            glassColor += lights[i].color * specularCoeff * ((shininess + 2.f)/(2.f*M_PIf)) *
                    pow(fmaxf(dot(ray.direction, reflectedLightRay),0.f), shininess) * radiance;

        }
    }

    result.x += glassColor.x;
    result.y += glassColor.y;
    result.z += glassColor.z;

    // recursive reflections
    if(specularity > 0.0f && prd_radiance.depth < maxDepth)
    {
        prd_radiance.depth++;
        float maxLambda = 10000.0f;
        Ray reflectedRay = make_Ray(hitPoint,reflect(ray.direction,normal),radianceRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower, reflectedRay, prd_radiance);
        result = (1.0f-specularity) * result + prd_radiance.result * specularity;
    }

    // recursive refractions
    //TODO: IMPLEMENT STACK IN TRACE AND BEER'S LAW
    if(prd_radiance.depth < maxDepth)
    {
        prd_radiance.depth++;
        float maxLambda = 10000.0f;
        Ray refractedRay = make_Ray(hitPoint,refract(ray.direction,normal, 1.0f, refractiveIdx),radianceRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower, refractedRay, prd_radiance);
        result = (1.0f-specularity) * result + prd_radiance.result * specularity;
    }

    result.w = 1.0f;

    prd_radiance.result = result/lights.size();

}

/*!
 * \brief Refracts a ray (direction).
 *
 * Refracts a ray with two \class GlassMaterial objects, including total reflection, based on Snell's law
 *
 * \param ray_in The 3D ray direction that goes into the object
 * \param normal The 3D surface normal of the object that is entered
 * \param n1 The refractive index of the object, the ray is coming from
 * \param n2 The refractive index of the object, the ray is entering
 * \return The 3D ray direction that after getting refracted on the surface
 */
static __device__ float3 refract(const float3 &ray_in, const float3 &normal, float n1, float n2)
{
    float alpha = dot(normal, ray_in);
    if(alpha > 1)
        return reflect(ray_in, normal);
    else
    {
        float n = n1/n2;
        float cosI = -1.0f * dot(normal, ray_in);
        float cosT2 = 1.0f - n * n * (1.0f - cosI * cosI);
        //if(cosT2 > 0.0f)
        {
            float3 t = (n * ray_in + (n * cosI - sqrt(cosT2)) * normal);
            return t;
        }

    }
}
