#include "hip/hip_runtime.h"

#include "../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(optix::Ray, ray, rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float, intersectionDistance, rtIntersectionDistance,);
rtDeclareVariable(float3,color,,);
rtDeclareVariable(float3, normal, attribute normal,);

static __device__ void shadowed();
static __device__ void shade();

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}


/*! \fn shadowed
 * \brief method to generate a shadow
 *
 * Generate a new shadow ray if a the incoming ray hits an object
 */

static __device__ void shadowed()
{
    //create ray from hit to light
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}


/*! * \fn shade
 * \brief shade method for ray color
 * \var shadowPrd Shadow ray data (attenuation)
 * \var hitPoint Intersection coords with object
 * \var shadowDriection 3d vector of the shadow direction
 * \var shadowRay New ray to trace the shadow
 *
 * generate color, trace new shadow ray
 */

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;
    shadowPrd.attenuation = make_float3(1.0f);
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    for(unsigned int i = 0;i < lights.size();++i)
    {
        float3 hitPoint = ray.origin + intersectionDistance * ray.direction;
        float3 shadowDirection = lights[i].position - hitPoint;
        float maxLambda = length(shadowDirection);
        shadowDirection = normalize(shadowDirection);
        hitPoint = hitPoint + sceneEpsilon * shadowDirection;

        Ray shadowRay = make_Ray(hitPoint, shadowDirection,
                                 shadowRayType, sceneEpsilon,maxLambda);

        //trace new shadow ray
        rtTrace(topShadower, shadowRay, shadowPrd);
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            float4 ret = make_float4(color,1.f) * make_float4(lights[i].color,1.f);
            ret *= (lights[i].intensity * dot(normal,shadowDirection)) / (maxLambda * maxLambda);
            result += ret;
        }
    }
    result.w = 1.0f;
    prd_radiance.result = result/lights.size();
}

