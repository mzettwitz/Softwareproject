#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <math.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, anisotropicFactorU,,);
rtDeclareVariable(float, anisotropicFactorV,,);
rtDeclareVariable(float, diffuseCoeff,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3,geometricNormal,attribute geometricNormal,);
rtDeclareVariable(float3,shadingNormal,attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

static __device__ float3 orthoVector(float3 in);

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 n = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);
    n = normalize(n);

    float3 V = normalize(-ray.direction);

    float3 fr = make_float3(0,0,0);
    float3 irradiance = make_float3(0,0,0);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    float rd = diffuseCoeff;
    float rs = specularCoeff;

    float nU = anisotropicFactorU;
    float nV = anisotropicFactorV;

    // iterate over lights
    for(unsigned int i = 0;i < lights.size();++i)
    {
        shadowPrd.attenuation = make_float3(1.0f);

        // light values
        float3 L = lights[i].position - hitPoint;
        float maxLambda = length(L);
        L = normalize(L);

        float radiance = lights[i].intensity / (maxLambda * maxLambda);

        // offset
        hitPoint += (n * sceneEpsilon);

        Ray shadowRay = make_Ray(hitPoint,L,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        // fr
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            float3 h = (L + V);
            h = normalize(h);

            float VdotN = dot(V,n);
            float LdotN = dot(L,n);
            float LdotH = dot(L,h);

            // diffuse term pd
            float pd = ((28.f*rd)/(23.f*M_PIf))*(1.f-rs) * (1.f-powf(1.f-LdotN/2.f,5.f)) * (1.f-powf(1.f-VdotN/2.f,5.f));

            // Fresnel term F by Schlick's approximation
            float F = rs + (1.f - rs)*(powf(1.f-LdotH,5.f));

            // base vectors
            float3 u = orthoVector(n);
            u = normalize(u);
            float3 v = cross(n,u);
            v = normalize(v);

            float HdotU = dot(h,u);
            float HdotV = dot(h,v);
            float HdotN = dot(h,n);

            // specular term ps
            float ps1 = sqrtf((nU + 1.f) * (nV + 1.f)) / (8.f * M_PIf);
            float ps2 = powf(HdotN, (nU * HdotU * HdotU + nV * HdotV * HdotV)/(1.f - HdotN * HdotN));
            float ps3 = LdotH * fmaxf(LdotN, VdotN);

            float ps = ps1 * (ps2/ps3) * F;

            // final
            fr = color * pd + ps;
        }

        irradiance += fr * fmaxf(dot(n,L),0) * radiance * lights[i].color;
    }

    float4 result = make_float4(irradiance,1);

    prd_radiance.result = result;
}

// compute a vector, perpendicular to the input vector
static __device__ float3 orthoVector(float3 in)
{
    float3 out = make_float3(0,0,0);

    if(in.y == 0.f)         // y = 0
    {
        out.x = 1.f;
        if(in.z == 0.f)     // y = 0, z = 0
            return out;
        else
        {
            out.z = (in.x * out.x) / -in.z;
            return out;
        }
    }
    else if(in.z == 0.f)    // z = 0
    {
        out.x = 1.f;
        out.y = (in.x * out.x) / -in.y;
        return out;
    }
    else
    {
        out.y = 1.f;
        if(in.x == 0.f && in.z == 0.f)  // x = 0, z = 0
            return out;
        else
        {
            out.z = (in.y * out.y) / -in.z;
            return out;
        }
    }
}
