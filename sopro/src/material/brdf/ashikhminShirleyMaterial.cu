#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <math.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, anisotropicFactorU,,);
rtDeclareVariable(float, anisotropicFactorV,,);
rtDeclareVariable(float, diffuseCoeff,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3,geometricNormal,attribute geometricNormal,);
rtDeclareVariable(float3,shadingNormal,attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

static __device__ float3 orthoVector(float3 in);

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 N = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);

    float3 V = normalize(-ray.direction);

    float3 fr = make_float3(0,0,0);
    float3 irradiance = make_float3(0,0,0);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    for(unsigned int i = 0;i < lights.size();++i)
    {
        shadowPrd.attenuation = make_float3(1.0f);

        float3 L = lights[i].position - hitPoint;
        float maxLambda = length(L);
        L = normalize(L);

        float radiance = lights[i].intensity / (maxLambda * maxLambda);

        Ray shadowRay = make_Ray(hitPoint,L,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        // fr
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            float rd = diffuseCoeff;
            float rs = specularCoeff;

            float nU = anisotropicFactorU;
            float nV = anisotropicFactorV;

            float3 h = (L + V);
            h = normalize(h);

            float VdotN = dot(V,N);
            float LdotN = dot(L,N);
            float LdotH = dot(L,h);

            // diffuse term Kd
            float pd = ((28*rd)/(23*M_PIf))*(1.f-rs) * (1.f-powf(1.f-LdotN/2,5))*(1.f-powf(1.f-VdotN/2,5));

            // Fresnel term F by Schlick's approximation
            float F = rs + (1.f - rs)*(1.f-powf(LdotH,5));

            // base vectors
            float3 u = orthoVector(N);
            u = normalize(u);
            float3 v = cross(N,u);
            v = normalize(v);

            float HdotU = dot(h,u);
            float HdotV = dot(h,v);
            float HdotN = dot(h,N);

            float ps1 = sqrtf((nU+1.f)*(nV+1.f))/(8*M_PIf);
            float ps2 = powf(HdotN, (nU * HdotU * HdotU + nV * HdotV * HdotV)/(1.f - HdotN * HdotN));
            float ps3 = LdotH * fmaxf(LdotN, VdotN);

            float ps = ps1 * (ps2/ps3) * F;

            // final
            fr = color * pd + ps;
        }

        irradiance += fr * fmaxf(dot(N,L),0) * radiance * lights[i].color;
    }

    float4 result = make_float4(irradiance,1);

    prd_radiance.result = result;
}

// compute a vector, perpendicular to the input vector
static __device__ float3 orthoVector(float3 in)
{
    float3 out = make_float3(0,0,0);

    if(in.y == 0.f)         // y = 0
    {
        out.x = 1.f;
        if(in.z == 0.f)     // y = 0, z = 0
            return out;
        else
        {
            out.z = (in.x * out.x) / -in.z;
            return out;
        }
    }
    else if(in.z == 0.f)    // z = 0
    {
        out.x = 1.f;
        out.y = (in.x * out.x) / -in.y;
        return out;
    }
    else
    {
        out.y = 1.f;
        if(in.x == 0.f && in.z == 0.f)  // x = 0, z = 0
            return out;
        else
        {
            out.z = (in.y * out.y) / -in.z;
            return out;
        }
    }
}
