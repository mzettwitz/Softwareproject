#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <math.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, anisotropicFactorU,,);
rtDeclareVariable(float, anisotropicFactorV,,);
rtDeclareVariable(float, diffuseCoeff,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3,geometricNormal,attribute geometricNormal,);
rtDeclareVariable(float3,shadingNormal,attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

static __device__ float3 orthoVector(float3 in);

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 n = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);
    n = normalize(n);

    float3 V = normalize(-ray.direction);

    float3 fr = make_float3(0,0,0);
    float3 irradiance = make_float3(0,0,0);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    float ps = specularCoeff;

    float alphaX = anisotropicFactorU;
    float alphaY = anisotropicFactorV;

    // iterate over lights
    for(unsigned int i = 0;i < lights.size();++i)
    {
        shadowPrd.attenuation = make_float3(1.0f);

        // light values
        float3 L = lights[i].position - hitPoint;
        float maxLambda = length(L);
        L = normalize(L);

        float radiance = lights[i].intensity / (maxLambda * maxLambda);

        // offset
        hitPoint += n * sceneEpsilon;

        Ray shadowRay = make_Ray(hitPoint,L,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        // fr
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            float3 h = (L + V);
            float3 x = orthoVector(n);
            x = normalize(x);
            float3 y = cross(n,x);
            y = normalize(y);


            // diffuse term kd
            float kd = diffuseCoeff / M_PIf;

            // specular term ks
            // refered to Moroder-Dür
            float ks = 0;

            float HdotX = dot(h,x);
            float HdotY = dot(h,y);

            float factor1 =  (1.f/(M_PIf*alphaX*alphaY));
            float factor2 = (-1.f/powf(dot(h, n),2));
            float factor3 = (HdotX/alphaX) * (HdotX/alphaX);
            float factor4 = (HdotY/alphaY) * (HdotY/alphaY);

            h = normalize(h);

            float factor5 = 1.f/(4*powf(dot(L,h),2)*powf(dot(h,n),4));

            float specRef = factor1 * expf(factor2 * (factor3+factor4))*factor5;

            if (specRef > 1e-10f)
                ks = ps * specRef;

            // final
            fr = color * kd + ks;
        }

        irradiance += (fr * fmaxf(dot(n,L),0) * radiance * lights[i].color) * shadowPrd.attenuation;
    }

    float4 result = make_float4(irradiance,1);

    prd_radiance.result = result;
}

// compute a vector, perpendicular to the input vector
static __device__ float3 orthoVector(float3 in)
{
    float3 out = make_float3(0,0,0);

    if(in.y == 0.f)         // y = 0
    {
        out.x = 1.f;
        if(in.z == 0.f)     // y = 0, z = 0
            return out;
        else
        {
            out.z = (in.x * out.x) / -in.z;
            return out;
        }
    }
    else if(in.z == 0.f)    // z = 0
    {
        out.x = 1.f;
        out.y = (in.x * out.x) / -in.y;
        return out;
    }
    else
    {
        out.y = 1.f;
        if(in.x == 0.f && in.z == 0.f)  // x = 0, z = 0
            return out;
        else
        {
            out.z = (in.y * out.y) / -in.z;
            return out;
        }
    }
}
