#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, anisotropicFactorU,,);
rtDeclareVariable(float, anisotropicFactorV,,);
rtDeclareVariable(float, diffuseCoeff,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3,geometricNormal,attribute geometricNormal,);
rtDeclareVariable(float3,shadingNormal,attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 N = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);

    float3 V = normalize(-ray.direction);

    float3 Ks = make_float3(0,0,0);
    float3 Kd = make_float3(0,0,0);

    float3 fr =make_float3(0,0,0);
    float3 irradiance = make_float3(0,0,0);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    for(unsigned int i = 0;i < lights.size();++i)
    {
        fr = make_float3(0,0,0);
        shadowPrd.attenuation = make_float3(1.0f);

        float3 L = lights[i].position - hitPoint;
        float maxLambda = length(L);
        L = normalize(L);

        float3 H = (L + V) / length(L + V);
        H = normalize(H);

        float radiance = lights[i].intensity / (maxLambda * maxLambda);

        Ray shadowRay = make_Ray(hitPoint,L,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        //F fresnel term
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            Kd = diffuseCoeff * color / M_PI;

            float NdotL = dot(N,L);
            float NdotV = dot(N,V);
            float NdotH = dot(N,H);
            float3 X = make_float3(1,0,0);         //NOT CORRECT
            float3 Y = make_float3(0,1,0);         //NOT CORRECT

            float HdotX = dot(H,X);
            float HdotY = dot(H,Y);

            float ks1 = 1/(sqrt(NdotL * NdotV));
            float ks2 = NdotL / (4 * M_PIf * anisotropicFactorU * anisotropicFactorV);
            float ks3 = HdotX / anisotropicFactorU;
            ks3 *= ks3;
            float ks4 = HdotY/ anisotropicFactorV;
            ks4 *= ks4;
            float ks5 = -2 * (ks3 + ks4)/(1 + NdotH);
            ks5 = exp(ks5);

            Ks = ks1 * ks2 * ks5 * color * specularCoeff;

            fr = Kd + Ks;
        }

        irradiance += fr * fmaxf(dot(N,L),0) * radiance * lights[i].color;
    }

    irradiance = irradiance/lights.size();

    float4 result = make_float4(irradiance,1);

    prd_radiance.result = result;
}
