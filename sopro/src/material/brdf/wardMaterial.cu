#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, anisotropicFactorU,,);
rtDeclareVariable(float, anisotropicFactorV,,);
rtDeclareVariable(float, diffuseCoeff,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3,geometricNormal,attribute geometricNormal,);
rtDeclareVariable(float3,shadingNormal,attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

static __device__ float3 orthoVector(float3 in);

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 N = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);

    float3 V = normalize(-ray.direction);

    float3 Ks = make_float3(0,0,0);
    float3 Kd = make_float3(0,0,0);

    float3 fr = make_float3(0,0,0);
    float3 irradiance = make_float3(0,0,0);

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    for(unsigned int i = 0;i < lights.size();++i)
    {
        shadowPrd.attenuation = make_float3(1.0f);

        float3 L = lights[i].position - hitPoint;
        float maxLambda = length(L);
        L = normalize(L);



        float radiance = lights[i].intensity / (maxLambda * maxLambda);

        Ray shadowRay = make_Ray(hitPoint,L,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        //F fresnel term
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            // approximation
            float3 H = (L + V);
            H = normalize(H);

            // first term
            float ks = specularCoeff;

            float alphaX = anisotropicFactorU;
            float alphaY = anisotropicFactorV;

            float VdotN = dot(V,N);
            float LdotN = dot(L,N);

            float ks1 = ks/(4.f * M_PIf * alphaX * alphaY * sqrtf(VdotN*LdotN));

            // second term
            float3 X = orthoVector(N);
            X = normalize(X);
            float3 Y = cross(N,X);
            Y = normalize(Y);

            float HdotX = dot(H,X);
            float HdotY = dot(H,Y);
            float HdotN = dot(H,N);

            float HdX_aX_2 = (HdotX/alphaX) * (HdotX/alphaX);
            float HdY_aY_2 = (HdotY/alphaY) * (HdotY/alphaY);

            float ks2 = (HdX_aX_2 + HdY_aY_2) / (1 + HdotN);
            ks2 = -2.f*ks2;

            // final
            fr = diffuseCoeff * color / M_PI + ks1 * powf(M_Ef, ks2);
        }

        irradiance += fr * fmaxf(dot(N,L),0) * radiance * lights[i].color;
    }

    float4 result = make_float4(irradiance,1);

    prd_radiance.result = result;
}

// compute a vector, perpendicular to the input vector
static __device__ float3 orthoVector(float3 in)
{
    float3 out = make_float3(0,0,0);

    if(in.y == 0.f)         // y = 0
    {
        out.x = 1.f;
        if(in.z == 0.f)     // y = 0, z = 0
            return out;
        else
        {
            out.z = (in.x * out.x) / -in.z;
            return out;
        }
    }
    else if(in.z == 0.f)    // z = 0
    {
        out.x = 1.f;
        out.y = (in.x * out.x) / -in.y;
        return out;
    }
    else
    {
        out.y = 1.f;
        if(in.x == 0.f && in.z == 0.f)  // x = 0, z = 0
            return out;
        else
        {
            out.z = (in.y * out.y) / -in.z;
            return out;
        }
    }
}
