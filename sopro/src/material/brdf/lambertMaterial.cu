#include "hip/hip_runtime.h"

#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(optix::Ray, ray, rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float, intersectionDistance, rtIntersectionDistance,);
rtDeclareVariable(float3,color,,);
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal,);
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();
static __device__ void simpleShade();

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{

        shade();

}


/*! \fn shadowed
 * \brief method to generate a shadow
 *
 * Generate a new shadow ray if a the incoming ray hits an object
 */

static __device__ void shadowed()
{
    //create ray from hit to light
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}


/*! * \fn shade
 * \brief shade method for ray color
 * \var shadowPrd Shadow ray data (attenuation)
 * \var hitPoint Intersection coords with object
 * \var shadowDriection 3d vector of the shadow direction
 * \var shadowRay New ray to trace the shadow
 *
 * generate color, trace new shadow ray
 */

static __device__ void shade()
{


    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 N = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);
    //payload for shadow Ray
    PerRayData_shadow shadowPrd;
    //result color
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    float3 irradiance = make_float3(0,0,0);
    float3 fr = make_float3(0.0f,0.0f,0.0f);


    //iterate over every light source
    for(unsigned int i = 0;i < lights.size();++i)
    {
        fr = make_float3(0,0,0);
        shadowPrd.attenuation = make_float3(1.0f);

        //light values
        //get hitpoint on geometry
        float3 hitPoint = ray.origin + intersectionDistance * ray.direction;
        //get direction from hitpoint to lightsource
        float3 L = lights[i].position - hitPoint;
        float maxLambda = length(L);
        float radiance = lights[i].intensity / (maxLambda * maxLambda);
        L = normalize(L);

        //add sceneEpsilon
        hitPoint = hitPoint + sceneEpsilon * N;

        Ray shadowRay = make_Ray(hitPoint, L,
                                 shadowRayType, sceneEpsilon,maxLambda);

        //trace new shadow ray
        rtTrace(topShadower, shadowRay, shadowPrd);

        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
            fr = color / M_PIf;
        }

        irradiance += fmaxf(dot(N,L),0) * fr * lights[i].color * radiance;

    }
    result = make_float4(irradiance,1);
    prd_radiance.result = result;
}
