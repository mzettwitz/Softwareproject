#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;


rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int,maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, diffuseCoeff,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float, metallic,,);
rtDeclareVariable(float,subsurface,,);
rtDeclareVariable(float,specular,,);
rtDeclareVariable(float,specualarTint,,);
rtDeclareVariable(float,roughness,,);
rtDeclareVariable(float, anisotropic,,);
rtDeclareVariable(float, sheen,,);
rtDeclareVariable(float, sheenTint,,);
rtDeclareVariable(float, clearcoat,,);
rtDeclareVariable(float, clearcoatGloss,,);
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal,);
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not using \fn shadowed.
 */
RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

/*!
 * \brief Computes the closest intersection between camera ray and the \class SceneObject using \fn shade.
 */
RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

/*!
 * \brief Computes the color of the closest intersection point from camera and object.
 *
 * Basicly we set up a resulting color. This color is basicly black, it gains more color information with each
 * computation for each light source in the scene. \n
 * First we compute the hitpoint on the object. Now we iterate through each lightsource in the scene
 * and generate a new \class Ray from lightsource to the object to trace the shadow (using anyhit).
 * If there is shadow: add black, if there is no shadow, compute the phong based color.
 * Therefore we sum up the ambient light, the diffuse color and the specular color and add it,
 * weighted by the number of lights in the scene, to the resulting color.\n
 * At the end we generate and trace a new reflected \class Ray to compute the mirrored objects in the scene.
 *
 * \var shadowPrd The ray information for the shadow ray.
 * \var result RGBA color for the resulting color in the hitpoint.
 * \var p RGB color to sum up the three parts of the \class PhongMaterial
 * \var hitPoint 3D float vector for the intersection between camera ray and object
 * \var L 3D float vector for the direction of the actual lightsource
 * \var maxLambda Float value describing the distance between actual lightsource and hitpoint
 * \var R 3D vector for the direction of the light ray, reflected on the object surface
 * \var shadowRay A \class Ray to determine if the hitpoint is shadowed by the actual lightsource and any object in the scene
 * \var reflectedRay A \class Ray to trace the camera \class Ray that is reflected on the specular surface (mirror effect)
 * \var color4F RGBA version of the RGB diffuse color to give the reflection a color(lighted areas must be colored, since light is emitted)
 *
 */
static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 N = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);


    prd_radiance.result = make_float4(0.0f);
}

