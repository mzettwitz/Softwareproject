#include "hip/hip_runtime.h"
#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int, maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float,diffuseCoefficient,,);
rtDeclareVariable(float,specularCoefficient,,);
rtDeclareVariable(float,fresnelFactor,,);
rtDeclareVariable(float,roughness,,);
rtDeclareVariable(float,reflectance,,);
rtDeclareVariable(float3,geometricNormal,attribute geometricNormal,);
rtDeclareVariable(float3,shadingNormal,attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();

RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

RT_PROGRAM void closesthit_radiance()
{
    shade();
}

static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

static __device__ void shade()
{
    PerRayData_shadow shadowPrd;

    float3 geometricWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometricNormal));
    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));
    float3 N = faceforward(shadingWorldNormal,-ray.direction,geometricWorldNormal);

    float3 V = normalize(-ray.direction);

    float3 Ks = make_float3(0,0,0);
    float3 Kd = make_float3(0,0,0);

    float3 fr = make_float3(0,0,0);
    float3 irradiance = make_float3(0,0,0);
    //for specular term
    //Ks = color * (D *F *G) / (4 * VdotN * NdotL)
    float D = 0;
    float F = 0;
    float G = 0;

    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    for(unsigned int i = 0;i < lights.size();++i)
    {
        fr = make_float3(0,0,0);

        shadowPrd.attenuation = make_float3(1.0f);

        float3 L = lights[i].position - hitPoint;

        float maxLambda = length(L);
        L = normalize(L);

        float3 H = (L + V) / length(L+V);
        H = normalize(H);
        float radiance = lights[i].intensity / (maxLambda * maxLambda);

        Ray shadowRay = make_Ray(hitPoint,L,shadowRayType,sceneEpsilon,maxLambda);
        rtTrace(topShadower,shadowRay,shadowPrd);

        //F fresnel term
        if(fmaxf(shadowPrd.attenuation) > 0.0f)
        {
        F = pow((1 + dot(V,N)),fresnelFactor);

        //G geometric attenuation

        float HdotN = dot(H,N);
        float VdotN = dot(V,N);
        float VdotH = dot(V,H);
        float LdotN = dot(L,N);

        float g1 = (2 * HdotN * VdotN)/VdotH;
        float g2 = (2 * HdotN * LdotN)/VdotH;

        G = fminf(1,fminf(g1,g2));

        //D Beckmann distribution

        float alpha = acos(HdotN);
        float cosSqalpha = cos(alpha) * cos(alpha);

        float d1 = (1-cosSqalpha)/(cosSqalpha* roughness * roughness);


        D = exp(-d1)/(M_PIf * roughness * roughness * cosSqalpha*cosSqalpha);

        Ks = (D * F * G)/(4 * VdotN * LdotN) * color * specularCoefficient;

        Kd = color * diffuseCoefficient / M_PI;

        fr += Kd + Ks;
        }

        irradiance += fr * fmaxf(dot(N,L),0) * radiance * lights[i].color;

    }

    float4 result = make_float4(irradiance,1);

    // recursive reflections
    if(reflectance > 0.0f && prd_radiance.depth < maxDepth)
    {

        PerRayData_radiance prd_radiance_reflect;
        prd_radiance_reflect.depth = prd_radiance.depth+1;

        float maxLambda = 10000.0f;
        Ray reflectedRay = make_Ray(hitPoint,reflect(ray.direction,N),radianceRayType,sceneEpsilon,maxLambda);
        rtTrace(topObject, reflectedRay, prd_radiance_reflect);
        result = (1.0f-reflectance) * result + prd_radiance_reflect.result * reflectance;
    }

    prd_radiance.result = result;
}
