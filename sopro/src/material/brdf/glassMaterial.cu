#include "hip/hip_runtime.h"
// TODO:
// - Beer's law

#include "../../../include/structs.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(Ray,ray,rtCurrentRay,);
rtDeclareVariable(unsigned int, shadowRayType,,);
rtDeclareVariable(unsigned int, radianceRayType,,);
rtDeclareVariable(float, sceneEpsilon,,);
rtDeclareVariable(rtObject, topShadower,,);
rtDeclareVariable(rtObject, topObject,,);
rtDeclareVariable(unsigned int,maxDepth,,);
rtBuffer<PointLight> lights;
rtDeclareVariable(float,intersectionDistance,rtIntersectionDistance,);
rtDeclareVariable(float3, color,,);
rtDeclareVariable(float, refractiveIdx,,);
rtDeclareVariable(float, shininess,,);
rtDeclareVariable(float, specularCoeff,,);
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal,);

static __device__ void shadowed();
static __device__ void shade();
static __device__ bool refract(const float3 &ray_in, const float3 &normal, float n1, float n2,float3 &T);

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not using \fn shadowed.
 */
RT_PROGRAM void anyhit_shadow()
{
    shadowed();
}

/*!
 * \brief Computes the closest intersection between camera ray and the \class SceneObject using \fn shade.
 */
RT_PROGRAM void closesthit_radiance()
{
    shade();
}

/*!
 * \brief Determines whether a shadow ray hits any object in the scene or not, sets the attenuation to 0 and terminates the ray.
 */
static __device__ void shadowed()
{
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

/*!
 * \brief Computes the color of the closest intersection point from camera and object.
 *
 * Basicly we set up a resulting color. This color is basicly black, it gains more color information with each
 * computation for each light source in the scene. \n
 * First we compute the hitpoint on the object. Now we iterate through each lightsource in the scene
 * and generate a new \class Ray from lightsource to the object to trace the shadow (using anyhit).
 * If there is shadow: add black, if there is no shadow, compute the reflected and refracted color.
 * Therefore we reflect(generate) some rays to the outside and refract(generate) some rays to the inside. The refracted Ones depend on the
 * refractive index using Snell's law and Beer's law. \n
 *
 * \var shadowPrd The ray information for the shadow ray.
 * \var result RGBA color for the resulting color in the hitpoint.
 * \var reflectiveColor RGB color for reflective color information in the hitpoint
 * \var refractiveColor RGB color for the refractive color information in the hintpoint
 * \var glassColor RGB color for the sum of the reflection, refraction and the specular light distribution(phong based)
 * \var hitPoint 3D float vector for the intersection between camera ray and object
 * \var lightDirection 3D float vector for the direction of the actual lightsource
 * \var maxLambda Float value describing the distance between actual lightsource and hitpoint
 * \var reflectedLightRay 3D vector for the direction of the light ray, reflected on the object surface
 * \var refractedLightRay 3D vector for the direction of the light ray, refracted on the object surface
 * \var shadowRay A \class Ray to determine if the hitpoint is shadowed by the actual lightsource and any object in the scene
 * \var reflectedRay A \class Ray to trace the camera \class Ray that is reflected on the specular surface (mirror effect)
 * \var refractedRay A \class Ray to trace the camera \class Ray that is refracted on the transparent surface (refraction effect)
 *
 */


static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

static __device__ void shade()
{

    //color information
    float4 result = make_float4(0.0f,0.0f,0.0f,1.0f);
    float4 specularColor = make_float4(0,0,0,1);

    PerRayData_shadow shadowPrd;

    //hitpoint information
    float3 hitPoint = ray.origin + intersectionDistance * ray.direction;

    float3 shadingWorldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shadingNormal));

    float3 D = ray.direction;
    float3 N = shadingWorldNormal;
    float3 R = make_float3(0,0,0);
    float3 T = make_float3(0,0,0);
    float3 offset1 = make_float3(1,1,1);
    float3 offset2 = make_float3(1,1,1);
    float  dotD = 0;

    bool totalReflection = false;

    float3 beer_attenuation;

    if(dot(N, D) > 0) {
      // Beer's law attenuation
            float3 extinction = make_float3(log(color.x),log(color.y),log(color.z));
      beer_attenuation = exp(extinction * intersectionDistance);
    } else {
      beer_attenuation = make_float3(1);
    }
    //Glass part
    //outside of object
    if(dot(D,N) <= 0)
    {
        R = reflect(D,N);
        offset1 = hitPoint + sceneEpsilon * N;
        refract(D,N,1,refractiveIdx,T);
        offset2 = hitPoint - sceneEpsilon * N;
        dotD = dot(-D,N);

        //add specular reflections

        for(unsigned int i = 0;i < lights.size();++i)
        {
            shadowPrd.attenuation = make_float3(1.0f);
            float3 lightDirection = lights[i].position - hitPoint;

            float maxLambda = length(lightDirection);

            float radiance = lights[i].intensity / (maxLambda * maxLambda);

            lightDirection = normalize(lightDirection);
            float3 reflectedLightRay = normalize(reflect(lightDirection,shadingWorldNormal));

            Ray shadowRay = make_Ray(hitPoint,lightDirection,shadowRayType,sceneEpsilon,maxLambda);
            rtTrace(topShadower,shadowRay,shadowPrd);

            if(fmaxf(shadowPrd.attenuation) > 0.0f)
            {
                specularColor = make_float4(lights[i].color * specularCoeff * ((shininess + 2.f)/(2.f*M_PIf)) *
                    pow(fmaxf(dot(ray.direction, reflectedLightRay),0.f), shininess) * radiance,1.0f);
            }
                    result += specularColor;
        }


    }
    //inside of object
    else
    {
        totalReflection = refract(D,-N,refractiveIdx,1,T);
        if(totalReflection)
        {
            dotD = dot(T,N);
            offset2 = hitPoint + sceneEpsilon * N;
        }
        else
        {
            R = reflect(D,-N);
            offset1 = hitPoint - sceneEpsilon * N;

            PerRayData_radiance prd_reflected;
            prd_reflected.depth = prd_radiance.depth+1;
            Ray reflectedRay = make_Ray(hitPoint,R,radianceRayType,sceneEpsilon,10000.0f);
            if(prd_reflected.depth < 10)
            {
                rtTrace(topObject,reflectedRay,prd_reflected);
                result = prd_reflected.result;
            }
            prd_radiance.result = result;
            return;
        }
    }

    float r0 = (1.f-refractiveIdx)/(1.f+refractiveIdx);
    r0 *= r0;
    float r1 = r0 + (1.f-r0) * (1.f-dotD)* (1.f-dotD)* (1.f-dotD)* (1.f-dotD)* (1.f-dotD);

    Ray reflectedRay = make_Ray(offset1,R,radianceRayType,sceneEpsilon,10000.0f);
    Ray refractedRay = make_Ray(offset2,T,radianceRayType,sceneEpsilon,10000.0f);

    PerRayData_radiance prd_reflected;
    prd_reflected.depth = prd_radiance.depth+1;
    prd_reflected.result = make_float4(0,0,0,0);
    PerRayData_radiance prd_refracted;
    prd_refracted.result = make_float4(0,0,0,0);
    prd_refracted.depth = prd_radiance.depth+1;

    if(prd_reflected.depth < maxDepth)
    {
        rtTrace(topObject,reflectedRay,prd_reflected);
    }
    if(prd_refracted.depth < maxDepth)
    {
        rtTrace(topObject,refractedRay,prd_refracted);
    }
    result += ( r1 * prd_reflected.result + (1-r1) * prd_refracted.result) * make_float4(beer_attenuation,1.f);




    result.w = 1.0f;

    prd_radiance.result = result;

}

/*!
 * \brief Refracts a ray (direction).
 *
 * Refracts a ray with two \class GlassMaterial objects, including total reflection, based on Snell's law
 *
 * \param D The 3D ray direction that goes into the object
 * \param N The 3D surface normal of the object that is entered
 * \param n1 The refractive index of the object, the ray is coming from
 * \param n2 The refractive index of the object, the ray is entering
 * \param T The refracted Ray that goes through the object
 * \return The 3D ray direction that after getting refracted on the surface
 */
static __device__ bool refract(const float3 &D, const float3 &N, float n1, float n2,float3 &T)
{
    float d = (1 - ((n1*n1 * (1- dot(D,N)*dot(D,N))) / (n2*n2)));
    if(d >= 0)
    {
        T = D - N * dot(D,N) * (n1/n2) - N * sqrt(d);
        return true;
    }


    return false;
}
